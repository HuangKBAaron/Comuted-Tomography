#include "hip/hip_runtime.h"
#include<Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>
#include <stdio.h>
#include<time.h>
#include <iostream>
#include<device_atomic_functions.h>
#include "opencv2/opencv.hpp"
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime_api.h>
#include<stdlib.h>
#include<hip/device_functions.h>
#include "opencv2\core\core.hpp"

#include<fstream>
#include<algorithm>
#define PATCH 5
#define WINDOW 10
#define IMG_CNT 5
#define usg unsigned char 

using namespace cv;
using namespace std;

typedef struct point{
	int sum, image, x, y;
}point;

boolean comparator(const point &p1, const point &p2)
{
	if (p1.sum < p2.sum)
		return true;
	else if (p1.sum>p2.sum)
		return false;
	else return p1.image < p2.image;
}
__global__ void calc_diff(usg *cuda_s, usg *cuda_o, point sol[][102][25*IMG_CNT], int rows, int cols) {
	sol[blockIdx.y][blockIdx.x][threadIdx.x].sum = 99999999;
	sol[blockIdx.y][blockIdx.x][threadIdx.x].image = -1;
	sol[blockIdx.y][blockIdx.x][threadIdx.x].x = -1;
	sol[blockIdx.y][blockIdx.x][threadIdx.x].y = -1;
	int x = blockIdx.y * 5;
	int y = blockIdx.x * 5;
	int image_index = threadIdx.x % 25;
	int x_test = x - 10 + (image_index / 5) * 5;
	int y_test = y - 10 + (image_index % 5) * 5;

	if (x_test >= 0 && x_test < rows - 5 && y_test >= 0 && y_test < cols - 5 && x >= 0 && x < rows - 5 && y >= 0 && y < cols - 5) {
		int sum = 0;
		for (int m = 0; m < PATCH; m++)
			for (int n = 0; n < PATCH; n++)
				sum += ((int)cuda_s[(x + m)*cols + y + n] - (int)cuda_o[(threadIdx.x / 25) * 512 * 512 + (x_test + m)*cols + y_test + n])*((int)cuda_s[(x + m)*cols + y + n] - (int)cuda_o[(threadIdx.x / 25) * 512 * 512 + (x_test + m)*cols + y_test + n]);
		sol[blockIdx.y][blockIdx.x][threadIdx.x].sum = sum;
		sol[blockIdx.y][blockIdx.x][threadIdx.x].image = threadIdx.x  / 25;
		sol[blockIdx.y][blockIdx.x][threadIdx.x].x = x_test;
		sol[blockIdx.y][blockIdx.x][threadIdx.x].y = y_test;
	}
	__syncthreads();
}

int main()
{
	LARGE_INTEGER frequency; // ticks per second
	LARGE_INTEGER t1, t2; // ticks
	double elapsedTime;

	Mat subject = imread("C:\\Users\\User\\Downloads\\subject.jpg", CV_LOAD_IMAGE_GRAYSCALE);
	Mat other[5];
	other[0] = imread("C:\\Users\\User\\Downloads\\sample images\\1.png", CV_LOAD_IMAGE_GRAYSCALE);
	other[1] = imread("C:\\Users\\User\\Downloads\\sample images\\2.png", CV_LOAD_IMAGE_GRAYSCALE);
	other[2] = imread("C:\\Users\\User\\Downloads\\sample images\\3.png", CV_LOAD_IMAGE_GRAYSCALE);
	other[3] = imread("C:\\Users\\User\\Downloads\\sample images\\4.png", CV_LOAD_IMAGE_GRAYSCALE);
	other[4] = imread("C:\\Users\\User\\Downloads\\sample images\\5.png", CV_LOAD_IMAGE_GRAYSCALE);
	

	if (!subject.data)
	{
		cout << "Subject Image not found\nTerminaion process\n";
		return 0;

	}

	for (int i = 0; i < IMG_CNT; i++)
		if (!other[i].data)
			cout << "Image cant be loaded " << i << "\n";

	usg *cuda_s, *cuda_o, *s, *o;

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&t1);
	typedef point  narray[102][25*IMG_CNT];
	narray  *cuda_sol, *sol;
	sol = (narray *)malloc(102 * 102 * 25 * IMG_CNT * sizeof(point));

	hipMalloc((void **)&cuda_sol, (102 * 102 * 25 * IMG_CNT)*sizeof(point));

	s = (usg *)malloc(sizeof(char)*subject.rows*subject.cols);
	o = (usg *)malloc(sizeof(char) * 512 * 512 * IMG_CNT);

	hipMalloc((void **)&cuda_s, sizeof(char)*subject.rows*subject.cols);


		hipMalloc((void **)&cuda_o, sizeof(char)*512*512*IMG_CNT);

	for (int i = 0; i < subject.rows*subject.cols; i++)
		s[i] = subject.at<uchar>(i / subject.cols, i%subject.cols);
	for (int k = 0; k < IMG_CNT; k++)
		for (int i = 0; i < other[k].rows*other[k].cols; i++)
			o[k * 512 * 512 + i] = other[k].at<uchar>(i / other[k].cols, i%other[k].cols);

	hipMemcpy(cuda_s, s, sizeof(char)*subject.rows*subject.cols, hipMemcpyHostToDevice);
	hipMemcpy(cuda_o, o, sizeof(char)*512*512*IMG_CNT, hipMemcpyHostToDevice);

	ofstream myfile;
	myfile.open("C:\\Users\\User\\Downloads\\Prior_study\\game_cuda.txt");

	dim3 blockDim(25 * IMG_CNT);
	dim3 gridDim(102,102);
	
	
	calc_diff << <gridDim, blockDim >> > (cuda_s, cuda_o, cuda_sol, subject.rows, subject.cols);
	hipMemcpy(sol, cuda_sol, 102 * 102 * 25 * IMG_CNT * sizeof(point), hipMemcpyDeviceToHost);


	QueryPerformanceCounter(&t2);
	// compute and print the elapsed time in millisec
	elapsedTime = (t2.QuadPart - t1.QuadPart) * 1000.0 /
		frequency.QuadPart;

	for (int i = 0; i < 102; i++)
		for (int j = 0; j < 102; j++){
			sort(sol[i][j], sol[i][j] + 25*IMG_CNT,&comparator);
			
				myfile <<"subject image row = "<<i*5<<" "<<"subject image column = "<<5*j<<" "<< "Difference = "<<(int)sol[i][j][0].sum << " " << "otherimage number = "<<(int)sol[i][j][0].image << " " << "other image row = "<<(int)sol[i][j][0].x << " " <<"other image column = "<< (int)sol[i][j][0].y << "\t";
			
			myfile << "\n\n\n";
		}

	cout << elapsedTime << " ms.\n";

	hipFree(cuda_s);
	hipFree(cuda_o);
	hipFree(cuda_sol);
	free(sol);
	free(s);
	free(o);
	waitKey(0);
	return 0;
}